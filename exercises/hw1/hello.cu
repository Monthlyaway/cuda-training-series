
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello()
{
	printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main()
{
	hello<<<5, 10>>>();
	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	// cudaDeviceSynchronize();
}
